#include "hip/hip_runtime.h"
/* Parallel Best Band Selection Algorithm */
/*
Max Bands searched: 45 -> Taking Approximately a Day or so to complete 
Value returned: 0.094479 
Band Returning Max: UNKNOWN 

author: Michael C Estwanick 
*/

#include <thrust/extrema.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <sys/timeb.h>


#define ARRAYSIZE 45 //Number of Bands to check 
#define TOTAL powf(2,ARRAYSIZE)

//Cuda Error Check
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Kernel that Performs the best band selection algorithm 
__global__ void kernel(float *cc, long long int jump, float threadCount){

    
    int N = threadCount*threadCount; // Total threads
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int threadId = col + row * N; // Two dimensional Thread Index 

    int decimalNumber,quotient;
    int binaryNumber[ARRAYSIZE]; //Holds the binary number in an array

    //Spectra Data Set
    int a[169] = {1192, 1315, 1462, 1484, 1476, 1443, 1508, 1489, 1470, 1537, 1633, 1539, 1600, 1707, 1701, 1682, 1688, 1681, 1694, 1728, 1786, 1821, 1830, 1881, 1893, 1816, 1692, 1675, 1651, 1579, 1514, 1600, 1576, 1543, 1465, 1440, 1452, 1483, 1944, 2303, 2616, 3118, 3861, 4054, 3915, 4790, 5543, 4539, 4679, 5574, 5365, 5080, 4186, 4272, 4934, 5057, 5000, 4867, 3872, 2992, 2519, 1203, 1092, 1979, 3005, 3886, 4121, 4134, 4168, 4014, 3612, 3391, 2712, 1324, 473, 556, 1099, 1769, 1979, 2063, 2289, 2494, 2553, 2196, 2125, 2147, 1749, 1221, 667, 517, 732, 885, 988, 1051, 1001, 984, 997, 965, 1008, 1022, 992, 993, 982, 946, 850, 698, 562, 446, 334, 278, 226, 161, 99, 58, 125, 139, 101, 93, 115, 151, 167, 171, 178, 172, 180, 176, 163, 152, 143, 134, 129, 130, 139, 148, 151, 146, 137, 123, 106, 98, 79, 65, 58, 70, 60, 62, 51};

int b[169] = {1162, 1337, 1282, 1491, 1508, 1517, 1488, 1513, 1539, 1576, 1626, 1634, 1573, 1786, 1741, 1782, 1755, 1669, 1700, 1826, 1832, 1895, 1920, 1938, 1933, 1852, 1808, 1806, 1747, 1718, 1628, 1659, 1639, 1621, 1589, 1525, 1526, 1583, 2118, 2549, 2900, 3411, 4237, 4340, 4126, 4985, 5760, 4716, 4840, 5793, 5616, 5326, 4416, 4485, 5197, 5322, 5315, 5166, 4107, 3158, 2664, 1286, 1149, 2093, 3197, 4157, 4413, 4422, 4444, 4287, 3842, 3620, 2892, 1415, 498, 591, 1164, 1892, 2110, 2215, 2441, 2663, 2721, 2351, 2286, 2296, 1872, 1318, 714, 568, 805, 977, 1084, 1143, 1094, 1071, 1085, 1044, 1092, 1116, 1070, 1076, 1068, 1031, 928, 766, 617, 481, 370, 305, 250, 181, 108, 64, 139, 153, 109, 101, 122, 162, 180, 189, 192, 191, 195, 192, 178, 164, 153, 145, 141, 139, 148, 158, 163, 151, 148, 131, 120, 107, 91, 71, 72, 81, 65, 66, 62};

    //Holds Product from the dot product
    int c[ARRAYSIZE];
    //Arrays to hold integers summed 
    int aSumArr[ARRAYSIZE];
    int bSumArr[ARRAYSIZE];

    //Initialize arrays 
    for(int i = 0; i < ARRAYSIZE; i++){
        c[i] = 0;
        aSumArr[i] = 0;
        bSumArr[i] = 0;
        binaryNumber[i] = 0;
    }
                                                                                                                                                                                                                             
    
    int dotSum = 0; //value for the dot product
    int aSum = 0; //sum of valid array positions for array a
    int bSum = 0; //sum of valid array positions for array b
    int i = 0;
    float finalValue = 0; //Value of the arcCos of the dot product / sqrt(array a) * sqrt(array b)

    //Add jump to decimal to avoid running combinations that have already been calculated 
    decimalNumber = threadId + jump;
    quotient = decimalNumber;

    //Loop to convert decimal into binary and store in array
    while(quotient!=0){
        binaryNumber[i++]= quotient % 2;
        quotient = quotient / 2;
    }

    //Loop through binaryNumber array
    for(int x = ARRAYSIZE-1 ; x >= 0; x--){
        //Only perform calculation on selected bands
        if(binaryNumber[x] == 1){
            //Perform multiplication for dot product
            c[x] = a[x] * b[x];
            //Fill sum arrays at correct index
            aSumArr[x] = a[x];
            bSumArr[x] = b[x];
        }else{
            //Do Nothing
        }
    }

    //Sums up the product array to complete dot product
    for(int j = 0; j < ARRAYSIZE; ++j){
        dotSum += c[j]; // Dot Product 
        aSum += powf( aSumArr[j], 2 ); // Euclidean Norm on vector A
        bSum += powf( bSumArr[j], 2 ); // Euclidean Norm on vector B
    }

    //Create values for algorithm 
    float sqSum1 = sqrtf(aSum); //Finish Euclidean Norm on vector A
    float sqSum2 = sqrtf(bSum); //Finish Euclidean Norm on vector B
    float sqSum = sqSum1 * sqSum2; 
    float div = dotSum / sqSum ;
    //Plug in values for final answer
    finalValue = acosf( div ) ;

    //Stores the threads final value in array cc, in the respected index
    if(finalValue == finalValue){ //Check if the result is a real number 
        cc[threadId] = finalValue; //store value in array to be passed back to host (CPU)
    }else{
        cc[threadId] = -2; //If the value return is NaN set result = -2
    }
                                                                                                                                                                                                                             
}//End kernel 

float getFreeMem();
void deviceProperties();
float kernelCount(float freeMem, float totalMem);

int main( void ) {

    printf("------------------------------------------------------ \n");
    printf("2 ^ %d bands \n", ARRAYSIZE);
    hipDeviceReset();

    float freeMem = getFreeMem(); // Get available free memory
    float kernels = kernelCount( freeMem, TOTAL); // get number of kernels to launch 
    //Number of elements for each kernel
    float threadCount = ( TOTAL / kernels );
    printf("threadCount: Total thread Count: %lf \n", threadCount);
    //number of threads per kernel
    float threadsPerDim =  ceil( powf(threadCount,(.25f)) );
    printf("threadPerDim: Total threads per dimension: %lf \n", threadsPerDim);
    long long int jump = 0;
    
    float *h_c = (float *)malloc(sizeof(float)*threadCount); //Host Vector
    float *d_c; //Device Vector 
    //Collection of individual kernel max 
    float *maxCollection = (float *)malloc(sizeof(float)*kernels);

    float totalTime = 0.0;

    //CPU Timer start
    struct timeb start, end;
    int diff;
    ftime(&start);

    //Loop Through the kernel as many times needed to execute all bands,
    //When The GPU is out of memory the loop will execute again storing 
    //The max from each subset of bands in the maxCollection array
    for(int i = 0; i < kernels; i++){
        hipDeviceReset();
  
        //Setup Thread & Block Grid 
        dim3 blocks (threadsPerDim, threadsPerDim);
        dim3 threads (threadsPerDim, threadsPerDim);

        //Allocate Device Memory
        HANDLE_ERROR( hipMalloc((void**)&d_c, sizeof(float)*(threadCount)) );

        //Timer
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        
        //Execute Kernel
        kernel<<<blocks, threads>>>(d_c, jump, threadsPerDim);

        //Timer stuff
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        //printf("GPU: Kernel: %d Time:  %f \n", i ,milliseconds/1000);
        totalTime += milliseconds;

        //Retrieve vector from device holding the max value from each subset of bands
        HANDLE_ERROR( hipMemcpy(h_c, d_c, sizeof(float)*(threadCount), hipMemcpyDeviceToHost) );

        //Get the max value from the current subset of bands executed using THRUST Library 
        float *result = thrust::max_element(h_c, h_c + (int)threadCount);
        
        //Store the max in the maxCollection array
        maxCollection[i] = *result;
        //printf(" \t Jump Size: %ld \n", jump);
        jump = jump + threadCount; //Increment jump to avoid checking completed bands
        HANDLE_ERROR( hipFree(d_c) ); 

    }

    //Get max of all kernels 
    float *result = thrust::max_element(maxCollection, maxCollection + (int)kernels);
    //Print the maximum of all bands executed from all the kernels combined 
    printf("Total Max: is: %f \n", *result); 

    //Stop timer
    ftime(&end);
    diff = (int) (1000.0 * (end.time - start.time)
        + (end.millitm - start.millitm));

    printf("\nOperation took %u milliseconds\n", diff);
    //printf("Total GPU Time: %f \n", totalTime/1000 ); 

    return 0;
}
//Return the number of kernels 
float kernelCount(float freeMem, float totalMem){

    float totalSize = sizeof(float) * totalMem ;
    float kernels =  ceil( totalSize / freeMem ) ;

    printf("Total array size %lf || free mem %lf \n", totalSize, freeMem);
    printf("Kernels: %lf \n ", kernels);
    return kernels;
}
//Get available free memory 
float getFreeMem(){

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    //fprintf(stderr, "Free = %ld, Total = %ld\n", freeMem, totalMem);

    return freeMem;
}
//Get total memory of device 
void deviceProperties(){
    hipDeviceProp_t  prop;
    int devCount;
    HANDLE_ERROR( hipGetDeviceCount( &devCount ) );
    for (int i=0; i< devCount; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    }

        //Number of threads
}
                                                                                                                                                                                                                             
